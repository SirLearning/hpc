#include "hip/hip_runtime.h"
/*https://blog.imfing.com/2020/05/cuda-image-processing/
 * Convert RGB image to grayscale
 *
 * Load image with lodepng (https://github.com/lvandeve/lodepng)
 *
 * Sample image: lena.png
 *
 * Y = 0.2126*R + 0.7152*G + 0.0722*B
 *
 * */

#include <iostream>
#include <string>

#include "hip/hip_runtime.h"
#include "lodepng.h"

#define CHANNELS 4          // RGBA in PNG
#define BLOCK_SIZE 16       // Thread block size
__global__ void RGB2GrayKernel(uint8_t* input_image,
    uint8_t* output_image,
    int width, int height) {

    // input_image size: width*height*Channels
    // output_image size: width*height

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if ((row < height) && (col < width)) {
        // Offset in Grayscale image
        int offset = row * width + col;

        // Get RGB values
        uint8_t r = input_image[offset * CHANNELS + 0];
        uint8_t g = input_image[offset * CHANNELS + 1];
        uint8_t b = input_image[offset * CHANNELS + 2];

        // Convert to grayscale
        output_image[offset] = 0.2126f * r + 0.7152f * g + 0.0722f * b;
    }
}


/* Main */
int main(int argc, char* argv[]) {
    const char* input_file = argc > 1 ? argv[1] : "lena.png";

    // Variables
    std::vector<uint8_t> image;
    unsigned int width, height;

    // Load image
    unsigned error = lodepng::decode(image, width, height, input_file);

    std::cout << width << " x " << height << std::endl;

    // Allocate memory for CUDA device
    size_t mem_size = width * height * sizeof(uint8_t);
    uint8_t* dev_input, * dev_output;

    hipMalloc(&dev_input, mem_size * CHANNELS);
    hipMemcpy(dev_input, image.data(), mem_size * CHANNELS, hipMemcpyHostToDevice);

    hipMalloc(&dev_output, mem_size);

    // Invoke CUDA kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((width + BLOCK_SIZE - 1) / dimBlock.x, (height + BLOCK_SIZE - 1) / dimBlock.y);

    RGB2GrayKernel <<<dimGrid, dimBlock >>> (dev_input, dev_output, width, height);

    // Copy output from device
    uint8_t* image_y = new uint8_t[width * height];
    hipMemcpy(image_y, dev_output, mem_size, hipMemcpyDeviceToHost);

    hipFree(dev_input);
    hipFree(dev_output);

    // Prepare for output
    std::vector<uint8_t> out_image(image.size(), 255);
    for (size_t i = 0; i < width * height; i++) {
        size_t offset = i * CHANNELS;
        out_image[offset + 0] = out_image[offset + 1] = out_image[offset + 2] = image_y[i];
    }

    // Save processed image
    std::string output_file(input_file);
    output_file += ".gray.png";

    error = lodepng::encode(output_file, out_image, width, height);

    delete[] image_y;
    return 0;
}


