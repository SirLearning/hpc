#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>



const int NUM=1024*1024*32;

const int RADIUS = 3;
const int BLOCK_SIZE=256;
const int TIMES = 10;
__global__ void one_stencil(int n, const double *__restrict__ in, double *__restrict__ out){
int index=blockIdx.x * blockDim.x + threadIdx.x;
if(RADIUS <= index <= (n-RADIUS)){
  out[index] = (in[index - (RADIUS)] + in[index - RADIUS + 1] + in[index] + in[index + RADIUS - 1] + in[index + RADIUS])*0.1 + 2000;
 }
}

__global__ void one_stencil_shm(int n, const double *__restrict__ in, double *__restrict__ out){
  __shared__ double temp[BLOCK_SIZE + 2*RADIUS];
  int gindex=blockIdx.x * blockDim.x + threadIdx.x;
  if(gindex >= n){
   __syncthreads();
  return; 
  }
  int lindex = threadIdx.x + RADIUS;
  temp[lindex] = in[gindex];
  if (threadIdx.x < RADIUS) {  // fill in halos
    if(RADIUS <= gindex <=(n-RADIUS)){
      temp[lindex - RADIUS] = in[gindex - RADIUS];
      temp[lindex + BLOCK_SIZE] = in[gindex + BLOCK_SIZE];
    }
  }
  __syncthreads();
  double result = 0.0;
if(RADIUS <= gindex <= (n-RADIUS)){
  result = (temp[lindex - (RADIUS)] + temp[lindex - RADIUS + 1] + temp[lindex] + temp[lindex + RADIUS - 1] + temp[lindex + RADIUS])*0.1 + 2000;
  out[gindex]=result; 
  }
 
 return; 
}
void fill_array(const int n, double *array){
double init = (rand()%1000) * 0.2;
for(int ii=0; ii < n; ++ii){
*(array + ii) = init + ii * 0.00001;
}
}

inline int64_t GetUsec() {
  struct timeval tv;
  gettimeofday(&tv, NULL);
  return (tv.tv_sec * 1000000l) + tv.tv_usec;
}

int main(){
 srand(202404);
 double *host_x = new double[NUM];
 double *host_y = new double[NUM];
 fill_array(NUM, host_x);
 fill_array(NUM, host_y);

 hipEvent_t start, stop;
 hipEventCreate(&start);
 hipEventCreate(&stop);
 
 double *dev_x=nullptr, *dev_y=nullptr;
 hipError_t result=hipMalloc(&dev_x, sizeof(double)*(size_t)NUM);
 printf("gpu address=%p\n",dev_x);
 result = hipMalloc(&dev_y, sizeof(double)*(size_t)NUM); 
 printf("gpu address=%p\n",dev_y);
 hipMemcpy(dev_x, host_x, sizeof(double)*(size_t)NUM, hipMemcpyHostToDevice); 
 hipMemcpy(dev_y, host_y, sizeof(double)*(size_t)NUM, hipMemcpyHostToDevice); 
 
 int numBlocks = (NUM + BLOCK_SIZE - 1)/BLOCK_SIZE; 
 printf("numBlocks=%d\n",numBlocks);fflush(stdout);
 one_stencil<<<dim3(numBlocks,1,1),dim3(BLOCK_SIZE,1,1)>>>(NUM,dev_x,dev_y);
 hipDeviceSynchronize();
 
 hipEventRecord(start);
 for(int loop = 0; loop < TIMES; ++loop){
 one_stencil<<<dim3(numBlocks,1,1),dim3(BLOCK_SIZE,1,1)>>>(NUM,dev_x,dev_y);
 }
 hipEventRecord(stop);
 //cudaDeviceSynchronize();
 hipEventSynchronize(stop);
 hipMemcpy(host_y, dev_y, sizeof(double)*NUM, hipMemcpyDeviceToHost); 
 float ms=0.0f;
 hipEventElapsedTime(&ms, start, stop);
 printf("NO_SHM kernel time=%.5f\n",ms/TIMES);fflush(stdout); 
 printf("NO_SHM results: host_y[2]=%.5f,host_y[1000]=%.5f,host_y[NUM-3]=%.5f\n",host_y[2],host_y[1000],host_y[NUM-3]);fflush(stdout); 

 one_stencil_shm<<<dim3(numBlocks,1,1),dim3(BLOCK_SIZE,1,1)>>>(NUM,dev_x,dev_y);
 hipDeviceSynchronize();

 hipEventRecord(start);
 for(int loop = 0; loop < TIMES; ++loop){
 one_stencil_shm<<<dim3(numBlocks,1,1),dim3(BLOCK_SIZE,1,1)>>>(NUM,dev_x,dev_y);
 }
 hipEventRecord(stop);
 //cudaDeviceSynchronize();
 hipEventSynchronize(stop);
 hipMemcpy(host_y, dev_y, sizeof(double)*NUM, hipMemcpyDeviceToHost); 
 
 hipEventElapsedTime(&ms, start, stop);
 printf("SHM kernel time=%.5f\n",ms/TIMES);fflush(stdout); 
 printf("SHM results host_y[2]=%.5f,host_y[1000]=%.5f,host_y[NUM-3]=%.5f\n",host_y[2],host_y[1000],host_y[NUM-3]);fflush(stdout); 
 return 0;
}
