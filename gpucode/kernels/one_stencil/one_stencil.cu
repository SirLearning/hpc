#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>



const int NUM=1024*1024*32;

const int RADIUS = 2;
const int blockSize=256;
const int TIMES = 10;
__global__ void one_stencil(int n, double *xx, double *yy){
int index=blockIdx.x * blockDim.x + threadIdx.x;
if(RADIUS <= index <= (n-RADIUS)){
  yy[index] = (xx[index - (RADIUS)] + xx[index - RADIUS + 1] + xx[index] + xx[index + RADIUS - 1] + xx[index + RADIUS])*0.1 + 2000;
 }	
}

void fill_array(const int n, double *array){
double init = (rand()%1000) * 0.2;
for(int ii=0; ii < n; ++ii){
*(array + ii) = init + ii * 0.00001;
}
}

inline int64_t GetUsec() {
  struct timeval tv;
  gettimeofday(&tv, NULL);
  return (tv.tv_sec * 1000000l) + tv.tv_usec;
}

int main(){
 srand(202404);
 double *host_x = new double[NUM];
 double *host_y = new double[NUM];
 fill_array(NUM, host_x);
 fill_array(NUM, host_y);
 printf("host_y[1000]=%.5f\n",host_y[1000]);fflush(stdout); 

 hipEvent_t start, stop;
 hipEventCreate(&start);
 hipEventCreate(&stop);
 
 double *dev_x=nullptr, *dev_y=nullptr;
 hipError_t result=hipMalloc(&dev_x, sizeof(double)*(size_t)NUM);
 result = hipMalloc(&dev_y, sizeof(double)*(size_t)NUM); 
 printf("result=%d\n",result); 
 hipMemcpy(dev_x, host_x, sizeof(double)*(size_t)NUM, hipMemcpyHostToDevice); 
 hipMemcpy(dev_y, host_y, sizeof(double)*(size_t)NUM, hipMemcpyHostToDevice); 

 int numBlocks = (NUM + blockSize - 1)/blockSize; 
 printf("numBlocks=%d\n",numBlocks);fflush(stdout);
 one_stencil<<<dim3(numBlocks,1,1),dim3(blockSize,1,1)>>>(NUM,dev_x,dev_y);
 hipDeviceSynchronize();
 hipEventRecord(start);
 int64_t  ustart = GetUsec();
 for(int loop = 0; loop < TIMES; ++loop){
 one_stencil<<<dim3(numBlocks,1,1),dim3(blockSize,1,1)>>>(NUM,dev_x,dev_y);
 }
 hipEventRecord(stop);
 //cudaDeviceSynchronize();
 hipEventSynchronize(stop);
 int64_t ufinish = GetUsec(); 
 hipMemcpy(host_y, dev_y, sizeof(double)*NUM, hipMemcpyDeviceToHost); 
 float ms=0.0f;
 hipEventElapsedTime(&ms, start, stop);
 printf("kernel time=%.5f\n",ms/TIMES);fflush(stdout); 
 printf("kernel usec=%ld,host_y[1000]=%.5f,host_y[NUM-5]=%.5f\n",(ufinish - ustart)/TIMES,host_y[1000],host_y[NUM-5]);fflush(stdout); 
 return 0;
}
